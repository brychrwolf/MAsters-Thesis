#include <string>

#include "cudaTimer.cuh"

CudaTimer::CudaTimer(){
	name = "unnamed";
	hipEventCreate(&startTime);
	hipEventCreate(&stopTime);
}

CudaTimer::CudaTimer(std::string s){
	name = s;
	hipEventCreate(&startTime);
	hipEventCreate(&stopTime);
}

CudaTimer::~CudaTimer(){
	hipEventDestroy(startTime);
	hipEventDestroy(stopTime);
}

std::string CudaTimer::getName(){
	return name;
}

void CudaTimer::start(){
	hipEventRecord(startTime);
}

void CudaTimer::stop(){
	hipEventRecord(stopTime);
}

float CudaTimer::getElapsedTime(){
	float elapsedTime;
	hipEventSynchronize(startTime);
	hipEventSynchronize(stopTime);
	hipEventElapsedTime(&elapsedTime, startTime, stopTime);
	return elapsedTime;
}

