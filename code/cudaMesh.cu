#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <random>
#include <set>
#include <string>
#include <vector>

#include "cudaMesh.cuh"
#include "cudaAccess.cuh"

//TODO: Only loads PLY files, should support other file types!

CudaMesh::CudaMesh(){
	//TODO: implement
}

CudaMesh::CudaMesh(CudaAccess* acc){
	ca = acc;
}

CudaMesh::~CudaMesh(){
}



/* Getters and Setters */
int CudaMesh::getNumVertices(){
	return numVertices;
}

int CudaMesh::getNumFaces(){
	return numFaces;
}

double* CudaMesh::getVertices(){
	return vertices;
}

double* CudaMesh::getFeatureVectors(){
	return featureVectors;
}

int* CudaMesh::getFaces(){
	return faces;
}

std::vector<std::set<int>> CudaMesh::getAdjacentVertices(){
	return adjacentVertices;
}

std::vector<std::set<int>> CudaMesh::getFacesOfVertices(){
	return facesOfVertices;
}

int* CudaMesh::getAdjacentVertices_runLength(){
	return adjacentVertices_runLength;
}

int* CudaMesh::getFacesOfVertices_runLength(){
	return facesOfVertices_runLength;
}

int CudaMesh::getNumAdjacentVertices(){
	return numAdjacentVertices;
}

int CudaMesh::getNumFacesOfVertices(){
	return numFacesOfVertices;
}

int* CudaMesh::getFlat_adjacentVertices(){
	return flat_adjacentVertices;
}

int* CudaMesh::getFlat_facesOfVertices(){
	return flat_facesOfVertices;
}

double* CudaMesh::getEdgeLengths(){
	return edgeLengths;
}

double* CudaMesh::getMinEdgeLength(){
	return minEdgeLength;
}

double* CudaMesh::getOneRingMeanFunctionValues(){
	return oneRingMeanFunctionValues;
}



void CudaMesh::setNumVertices(int upd){
	numVertices = upd;
}

void CudaMesh::setNumFaces(int upd){
	numFaces = upd;
}

void CudaMesh::setVertices(double* upd){
	vertices = upd;
}

void CudaMesh::setFeatureVectors(double* upd){
	featureVectors = upd;
}

void CudaMesh::setFaces(int* upd){
	faces = upd;
}

void CudaMesh::setAdjacentVertices(std::vector<std::set<int>> upd){
	adjacentVertices = upd;
}

void CudaMesh::setFacesOfVertices(std::vector<std::set<int>> upd){
	facesOfVertices = upd;
}

void CudaMesh::setAdjacentVertices_runLength(int* upd){
	adjacentVertices_runLength = upd;
}

void CudaMesh::setFacesOfVertices_runLength(int* upd){
	facesOfVertices_runLength = upd;
}

void CudaMesh::setNumAdjacentVertices(int upd){
	numAdjacentVertices = upd;
}

void CudaMesh::setNumFacesOfVertices(int upd){
	numFacesOfVertices = upd;
}

void CudaMesh::setFlat_adjacentVertices(int* upd){
	flat_adjacentVertices = upd;
}

void CudaMesh::setFlat_facesOfVertices(int* upd){
	flat_facesOfVertices = upd;
}

void CudaMesh::setEdgeLengths(double* upd){
	edgeLengths = upd;
}

void CudaMesh::setMinEdgeLength(double* upd){
	minEdgeLength = upd;
}

void CudaMesh::setOneRingMeanFunctionValues(double* upd){
	oneRingMeanFunctionValues = upd;
}



/* IO */
void CudaMesh::loadPLY(std::string fileName){
	bool inHeaderSection = true;
	int faceSectionBegin;
	int vi = 0;
	int fi = 0;
	
	int v_idx = 0;
	int x_idx;
	int y_idx;
	int z_idx;

	std::ifstream infile(fileName);
	
	// for Random Feature Vectors
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> dis(-1.0, 1.0);

	// read every line in the file
	std::string line;
	int lineNumber = 0;
	while(std::getline(infile, line)){
		// 3 sections: header, vertices, faces
		if(inHeaderSection){
			// parse for numVertices and numFaces
			if(line.substr(0, 7) == "element"){
				if(line.substr(8, 6) == "vertex"){
					std::vector<std::string> words = split<std::string>(line);
					std::istringstream convert(words[2]);
					convert >> numVertices;
				}else if(line.substr(8, 4) == "face"){
					std::vector<std::string> words = split<std::string>(line);
					std::istringstream convert(words[2]);
					convert >> numFaces;
				}
			// parse for coord indexes
			}else if(line.substr(0, 8) == "property"){
				std::vector<std::string> words = split<std::string>(line);
				if(words[2] == "x")
					x_idx = v_idx;
				else if(words[2] == "y")
					y_idx = v_idx;
				else if(words[2] == "z")
					z_idx = v_idx;
				v_idx++;
			}else if(line.substr(0, 10) == "end_header"){
				inHeaderSection = false;
				faceSectionBegin = lineNumber + 1 + numVertices;
				hipMallocManaged(&vertices, 3 * numVertices * sizeof(double));
				hipMallocManaged(&featureVectors, numVertices * sizeof(double));
				hipMallocManaged(&faces, 3 * numFaces * sizeof(int));
			}
		}else if(lineNumber < faceSectionBegin){
			std::vector<double> coords = split<double>(line);
			vertices[vi*3 + 0] = coords[x_idx];
			vertices[vi*3 + 1] = coords[y_idx];
			vertices[vi*3 + 2] = coords[z_idx];
			//TODO: Are feature vectors stored in PLY file? currently set to 1 or random
			featureVectors[vi] = dis(gen);//1;
			vi++;
		}else{
			std::vector<int> coords = split<int>(line);
			faces[fi*3 + 0] = coords[1]; //coords[0] is list size
			faces[fi*3 + 1] = coords[2];
			faces[fi*3 + 2] = coords[3];
			fi++;
		}
		lineNumber++;
	}
}



void CudaMesh::printMesh(){
	for(int v = 0; v < numVertices; v++){
		std::cout << "vertices[" << v << "] = ";
		for(int i=0; i < 3; i++){
			if(i > 0)
				std::cout << ", ";
			std::cout << vertices[v*3+i];
		}
		std::cout << " featureVector = " << featureVectors[v] << std::endl;
	}
	for(int f = 0; f < numFaces; f++)
		std::cout << f << " = {" << faces[f*3+0] << ", " << faces[f*3+1] << ", " << faces[f*3+2] << "}" <<std::endl;
}

void CudaMesh::printAdjacentVertices(){
	std::cerr << std::endl;
	for(int i = 0; i < numVertices; i++){
		std::cerr << "adjacentVertices[" << i << "] ";
		for(int elem : adjacentVertices[i])
			std::cerr << elem << " ";
		std::cerr << std::endl;
	}
}

void CudaMesh::printFacesOfVertices(){
	std::cerr << std::endl;
	for(int i = 0; i < numVertices; i++){
		std::cerr << "facesOfVertices[" << i << "] ";
		for(int elem : facesOfVertices[i])
			std::cerr << elem << " ";
		std::cerr << std::endl;
	}
}

void CudaMesh::printAdjacentVertices_RunLength(){
	std::cerr << std::endl;
	for(int i = 0; i < numVertices; i++){
		std::cerr << "adjacentVertices_runLength[" << i << "] " << adjacentVertices_runLength[i] << std::endl;
	}
}

void CudaMesh::printFacesOfVertices_RunLength(){
	std::cerr << std::endl;
	for(int i = 0; i < numVertices; i++){
		std::cerr << "facesOfVertices_runLength[" << i << "] " << facesOfVertices_runLength[i] << std::endl;
	}
}

void CudaMesh::printFlat_AdjacentVertices(){
	std::cerr << std::endl;
	for(int i = 0; i < numAdjacentVertices; i++){
		std::cerr << "flat_adjacentVertices[" << i << "] " << flat_adjacentVertices[i] << std::endl;
	}
}

void CudaMesh::printFlat_FacesOfVertices(){
	std::cerr << std::endl;
	for(int i = 0; i < numFacesOfVertices; i++){
		std::cerr << "flat_facesOfVertices[" << i << "] " << flat_facesOfVertices[i] << std::endl;
	}
}

void CudaMesh::printEdgeLengths(){
	std::cerr << std::endl;
	for(int i = 0; i < numAdjacentVertices; i++){
		std::cerr << "edgeLengths[" << i << "] " << edgeLengths[i] << std::endl;
	}
}

void CudaMesh::printMinEdgeLength(){
	std::cerr << std::endl;
	for(int i = 0; i < numVertices; i++){
		std::cerr << "minEdgeLength[" << i << "] " << minEdgeLength[i] << std::endl;
	}
}

void CudaMesh::printOneRingMeanFunctionValues(){
	std::cerr << std::endl;
	for(int i = 0; i < numVertices; i++){
		std::cerr << "oneRingMeanFunctionValues[" << i << "] " << oneRingMeanFunctionValues[i] << std::endl;
	}
}



/* Build Tables */
void CudaMesh::buildSets(){
	std::vector<std::set<int>>(numVertices).swap(adjacentVertices);
	std::vector<std::set<int>>(numVertices).swap(facesOfVertices);

	//TODO: Determine if this way is optimal:
	//	edges saved twice, once in each direction, but enables use of runLength array...
	for(int f = 0; f < numFaces; f++){
		for(int i = 0; i < 3; i++){ //TODO: relies on there always being 3 vertices to a face
			int a = f*3+(i+0)%3;
			int b = f*3+(i+1)%3;
			int c = f*3+(i+2)%3;
			int v = faces[a];
			adjacentVertices[v].insert(faces[b]);
			adjacentVertices[v].insert(faces[c]);
			facesOfVertices[v].insert(f);
		}
	}
}

void CudaMesh::determineRunLengths(){
	hipMallocManaged(&adjacentVertices_runLength, numVertices*sizeof(int));
	hipMallocManaged(&facesOfVertices_runLength,  numVertices*sizeof(int));
	
	std::cout << "Iterating over each vertex as v0..." << std::endl;
	adjacentVertices_runLength[0] = adjacentVertices[0].size();
	facesOfVertices_runLength[0]  = facesOfVertices[0].size();
	for(int v0 = 0+1; v0 < numVertices; v0++){
		adjacentVertices_runLength[v0] = adjacentVertices_runLength[v0-1] + adjacentVertices[v0].size();
		facesOfVertices_runLength[v0]  = facesOfVertices_runLength[v0-1]  + facesOfVertices[v0].size();
	}
	
	numAdjacentVertices = adjacentVertices_runLength[numVertices-1];
	numFacesOfVertices  = facesOfVertices_runLength[numVertices-1];
}

void CudaMesh::flattenSets(){
	hipMallocManaged(&flat_adjacentVertices, numAdjacentVertices*sizeof(int));
	hipMallocManaged(&flat_facesOfVertices, numFacesOfVertices*sizeof(int));

	int r = 0;
	int s = 0;
	for(int v0 = 0; v0 < numVertices; v0++){
		for(std::set<int>::iterator vi_iter = adjacentVertices[v0].begin(); vi_iter != adjacentVertices[v0].end(); vi_iter++){
			int vi = *vi_iter;
			flat_adjacentVertices[r] = vi;
			r++;
		}
		for(std::set<int>::iterator vi_iter = facesOfVertices[v0].begin(); vi_iter != facesOfVertices[v0].end(); vi_iter++){
			int vi = *vi_iter;
			flat_facesOfVertices[s] = vi;
			s++;
		}
	}
}

void CudaMesh::freeSets(){
	std::vector<std::set<int>>().swap(adjacentVertices);
	std::vector<std::set<int>>().swap(facesOfVertices);
}



/* Pre-Calculation */
void CudaMesh::preCalculateEdgeLengths(){
	hipMallocManaged(&edgeLengths, numAdjacentVertices*sizeof(double));
	int blockSize = (*ca).getIdealBlockSizeForProblemOfSize(numAdjacentVertices);
	int numBlocks = max(1, numAdjacentVertices / blockSize);
	std::cout << "getEdgeLengths<<<" << numBlocks << ", " << blockSize <<">>(" << numAdjacentVertices << ")" << std::endl;
	kernel_getEdgeLengths<<<numBlocks, blockSize>>>(numAdjacentVertices, numVertices, flat_adjacentVertices, adjacentVertices_runLength, vertices, edgeLengths);
	hipDeviceSynchronize();	//wait for GPU to finish before accessing on host
}

__global__
void kernel_getEdgeLengths(int numAdjacentVertices, int numVertices, int* flat_adjacentVertices, int* adjacentVertices_runLength, double* vertices, double* edgeLengths){
	//TODO Optimization analysis: storage vs speed
	//this:
	//	flat_adjacentVertices = 6nV (average 6 pairs per vertex)
	//	adjacentVertices_runLength = 1nV
	//	index search requires averagePairCount per Vertex (6nV)
	//fully indexed:
	//	flat_adjacentVertices = 3*6nV (can be halved if redundant AVs are not stored)
	//	no runLength required
	//	no index search time
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x; //0-95
	int stride = blockDim.x * gridDim.x; //32*3 = 96

	// Use all availble threads to do all numAdjacentVertices
	for(int av = global_threadIndex; av < numAdjacentVertices; av += stride){
		int vi = flat_adjacentVertices[av];
		int v0 = getV0FromRunLength(numVertices, av, adjacentVertices_runLength);
		edgeLengths[av] = cuda_l2norm_diff(vi, v0, vertices);
		//printf("edgeLength[%d]\t(v0 %d, vi %d)\t%g\n", av, v0, vi, edgeLengths[av]);
	}
}

__device__
int getV0FromRunLength(int numVertices, int av, int* adjacentVertices_runLength){
	//TODO: measure performance	
	//this: 
	//	pros, smaller memory, 
	//	cons, need this loop to determine v0! (do intelligent search instead)
	//alternatively: save v0 as a second value per index of flat_adjacentVertices
	//	pros, v0 is always known
	//	cons flat_adjacentVertices doubles in size
	int v0;
	for(int v = 0; v < numVertices; v++){
		if(av < adjacentVertices_runLength[v]){
			//printf("[%d, %d, %d, %d]:", blockIndex, local_threadIndex, global_threadIndex, av);
			v0 = v;
			break;
		}
	}
	return v0;
}

__device__
double cuda_l2norm_diff(int vi, int v0, double* vertices){
	// Too slow
	return sqrt((double) (vertices[(vi*3)+0] - vertices[(v0*3)+0])*(vertices[(vi*3)+0] - vertices[(v0*3)+0])
					   + (vertices[(vi*3)+1] - vertices[(v0*3)+1])*(vertices[(vi*3)+1] - vertices[(v0*3)+1])
					   + (vertices[(vi*3)+2] - vertices[(v0*3)+2])*(vertices[(vi*3)+2] - vertices[(v0*3)+2]));
	/* Even slower...!?
	int vi30 = (vi * 3);
	int vi31 = (vi * 3) + 1;
	int vi32 = (vi * 3) + 2;
	int v030 = (v0 * 3);
	int v031 = (v0 * 3) + 1;
	int v032 = (v0 * 3) + 2;
	return sqrt((double) (vertices[vi30] - vertices[v030]) * (vertices[vi30] - vertices[v030])
					   + (vertices[vi31] - vertices[v031]) * (vertices[vi31] - vertices[v031])
					   + (vertices[vi32] - vertices[v032]) * (vertices[vi32] - vertices[v032]));*/
}

void CudaMesh::preCalculateMinEdgeLength(){
	hipMallocManaged(&minEdgeLength, numVertices*sizeof(double));
	int blockSize = (*ca).getIdealBlockSizeForProblemOfSize(numVertices);
	int numBlocks = max(1, numVertices / blockSize);
	std::cout << "getMinEdgeLength<<<" << numBlocks << ", " << blockSize << ">>(" << numVertices << ")" << std::endl;
	kernel_getMinEdgeLength<<<numBlocks, blockSize>>>(numAdjacentVertices, numVertices, adjacentVertices_runLength, vertices, edgeLengths, minEdgeLength);
	hipDeviceSynchronize();
}

__global__
void kernel_getMinEdgeLength(int numAdjacentVertices, int numVertices, int* adjacentVertices_runLength, double* vertices, double* edgeLengths, double* minEdgeLength){
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	// Use all availble threads to do all numVertices as v0
	for(int v0 = global_threadIndex; v0 < numVertices; v0 += stride){
		int av_begin = (v0 == 0 ? 0 : adjacentVertices_runLength[v0-1]);
		for(int av = av_begin; av < adjacentVertices_runLength[v0]; av++){
			if(minEdgeLength[v0] <= 0 || edgeLengths[av] <= minEdgeLength[v0]){
				minEdgeLength[v0] = edgeLengths[av];
			}
		}
		//printf("minEdgeLength[%d] %f\n", v0, minEdgeLength[v0]);
	}
}

void CudaMesh::calculateOneRingMeanFunctionValues(){
	hipMallocManaged(&oneRingMeanFunctionValues, numVertices*sizeof(double));
	int blockSize = (*ca).getIdealBlockSizeForProblemOfSize(numVertices);
	int numBlocks = max(1, numVertices / blockSize);
	std::cout << "getOneRingMeanFunctionValues<<<" << numBlocks << ", " << blockSize << ">>(" << numVertices << ")" << std::endl;
	kernel_getOneRingMeanFunctionValues<<<numBlocks, blockSize>>>(
		numVertices, 
		adjacentVertices_runLength, 
		facesOfVertices_runLength, 
		flat_facesOfVertices, 
		flat_adjacentVertices, 
		faces,
		minEdgeLength,
		featureVectors,
		edgeLengths,
		oneRingMeanFunctionValues
	);
	hipDeviceSynchronize();
}

__global__
void kernel_getOneRingMeanFunctionValues(
	int numVertices, 
	int* adjacentVertices_runLength,
	int* facesOfVertices_runLength, 
	int* flat_facesOfVertices, 
	int* flat_adjacentVertices,
	int* faces, 
	double* minEdgeLength, 
	double* featureVectors, 
	double* edgeLengths,
	double* oneRingMeanFunctionValues
){
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x; //0-95
	int stride = blockDim.x * gridDim.x; //32*3 = 96

	double accuFuncVals = 0.0;
	double accuArea = 0.0;

	// Use all availble threads to do all numVertices as v0
	for(int v0 = global_threadIndex; v0 < numVertices; v0 += stride){
		int fi_begin = (v0 == 0 ? 0 : facesOfVertices_runLength[v0-1]);
		for(int fi = fi_begin; fi < facesOfVertices_runLength[v0]; fi++){
			//currFace->getFuncVal1RingSector( this, rMinDist, currArea, currFuncVal ); //ORS.307
				//get1RingSectorConst();
				int vi, vip1;
				getViAndVip1FromV0andFi(v0, flat_facesOfVertices[fi], faces, vi, vip1);
				//printf("[%d]\t[%d]\t%d\t%d\n", v0, flat_facesOfVertices[fi], vi, vip1);

				//TODO: Ensure edges A, B, C are correct with v0, vi, vip1; also regarding funcVals later
				//ORS.456
				double lengthEdgeA = getEdgeLengthOfV0AndVi(vi, vip1, adjacentVertices_runLength, flat_adjacentVertices, edgeLengths);
				double lengthEdgeB = getEdgeLengthOfV0AndVi(v0, vip1, adjacentVertices_runLength, flat_adjacentVertices, edgeLengths);
				double lengthEdgeC = getEdgeLengthOfV0AndVi(v0, vi,   adjacentVertices_runLength, flat_adjacentVertices, edgeLengths);
				double alpha = acos( ( lengthEdgeB*lengthEdgeB + lengthEdgeC*lengthEdgeC - lengthEdgeA*lengthEdgeA ) / ( 2*lengthEdgeB*lengthEdgeC ) );

				double rNormDist = minEdgeLength[v0];
				double lenCenterToA = lengthEdgeC;
				double lenCenterToB = lengthEdgeB;
			
				//ORS.403 Area - https://en.wikipedia.org/wiki/Circular_sector#Area
				//*changed from m to r to skip "passthrough" see ORS.372
				double rSectorArea = rNormDist * rNormDist * alpha / 2.0; // As alpha is already in radiant.

				//ORS.412 Function values interpolated f'_i and f'_{i+1}
				// Compute the third angle using alpha/2.0 and 90°:
				double beta = ( M_PI - alpha ) / 2.0;
				// Law of sines
				double diameterCircum = rNormDist / sin( beta ); // Constant ratio equal longest edge

				//ORS.420 Distances for interpolation
				double mRatioCA = diameterCircum / lenCenterToA;
				double mRatioCB = diameterCircum / lenCenterToB;
				// Circle segment, center of gravity - https://de.wikipedia.org/wiki/Geometrischer_Schwerpunkt#Kreisausschnitt
				double mCenterOfGravityDist = ( 2.0 * sin( alpha ) ) / ( 3.0 * alpha );

				//ORS.357 Fetch function values
				double funcValCenter = featureVectors[v0];
				double funcValA = featureVectors[vi];
				double funcValB = featureVectors[vip1];

				//ORS.365 Interpolate
				double funcValInterpolA = funcValCenter*(1.0-mRatioCA) + funcValA*mRatioCA;
				double funcValInterpolB = funcValCenter*(1.0-mRatioCB) + funcValB*mRatioCB;

				//ORS.369 Compute average function value at the center of gravity of the circle sector
				double rSectorFuncVal = funcValCenter*( 1.0 - mCenterOfGravityDist ) +
								 ( funcValInterpolA + funcValInterpolB ) * mCenterOfGravityDist / 2.0;

			double currFuncVal = rSectorFuncVal;
			double currArea = rSectorArea;
			
			//ORS.309
			accuFuncVals += currFuncVal * currArea;
			accuArea += currArea;
		}

		oneRingMeanFunctionValues[v0] = accuFuncVals / accuArea;
		//if(global_threadIndex % 1000 == 0)
		//	printf("oneRingMeanFunctionValues[%d] %f\n", v0, oneRingMeanFunctionValues[v0]);
	}
}

__device__
void getViAndVip1FromV0andFi(int v0, int fi, int* faces, int& vi, int& vip1){
	//printf("faces[%d*3+{0,1,2}] {%d, %d, %d}\n", fi, faces[(fi*3)+0], faces[(fi*3)+1], faces[(fi*3)+2]);
	bool isViAssigned = false;
	for(int i = 0; i < 3; i++){ // for each vertex in this face (a, b, c)
		int v = faces[fi*3+i];
		if(v != v0){ // exclude v0
			if(isViAssigned){
				vip1 = v; // assign the other corner to vip1
			}else{
				vi = v; // assign the first corner to vi
				isViAssigned = true;
			}
		}
	}
}

__device__
double getEdgeLengthOfV0AndVi(int v0, int vi, int* adjacentVertices_runLength, int* flat_adjacentVertices, double* edgeLengths){
	//TODO: Error handling?
	int av_begin = (v0 == 0 ? 0 : adjacentVertices_runLength[v0-1]);
	double edgeLength;
	for(int av = av_begin; av < adjacentVertices_runLength[v0]; av++){
		if(flat_adjacentVertices[av] == vi){
			edgeLength = edgeLengths[av];
			break;
		}
	}
	return edgeLength;
}

