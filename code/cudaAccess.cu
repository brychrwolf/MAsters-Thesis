#include <stdio.h>
#include "cudaAccess.h"

CudaAccess::CudaAccess(){
	printf("CudaAccess Constructed");	
}

void CudaAccess::printCUDAProps(int devCount){
	printf("There are %d CUDA devices.\n", devCount);

	// Iterate through devices
	for (int i = 0; i < devCount; ++i)
	{
		// Get device properties
		printf("\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		
    	printf("Name:                          %s\n",  devProp.name);
    	printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    	printf("Clock rate:                    %d\n",  devProp.clockRate);
    	printf("Total constant memory:         %u\n",  devProp.totalConstMem);
		printf("CUDA Capability Major/Minor version number:    %d.%d\n", devProp.major, devProp.minor);

	}
}

