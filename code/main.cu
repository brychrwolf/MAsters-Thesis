#include "hip/hip_runtime.h"
#include <stdio.h>

#include <array>
#include <cmath>
#include <cfloat>
#include <iostream>
#include <iterator>
#include <fstream>
#include <map>
#include <random>
#include <set>
#include <sstream>
#include <string>
#include <vector>

// to engage GPUs when installed in hybrid system, run as 
// optirun ./main

template<typename T>
std::vector<T> split(std::string line){
	std::istringstream iss(line);
	std::vector<T> results(std::istream_iterator<T>{iss},
						   std::istream_iterator<T>());
	return results;
}
template std::vector<std::string> split<std::string>(std::string);
template std::vector<int> split<int>(std::string);
template std::vector<float> split<float>(std::string);
template std::vector<double> split<double>(std::string);

void printCUDAProps(int devCount);
void loadMesh_ply(std::string fileName, int& numVertices, double** vertices, double** featureVectors, int& numFaces, int** faces);
void printMesh(int numVertices, double* vertices, double* featureVectors, int numFaces, int* faces);

__global__ void buildLookupTables(int numFaces, int* faces, int* facesOfVertices, int* adjacentVertices);
__global__ void getEdgeLengths(int numAdjacentVertices, int numVertices, int* flat_adjacentVertices, int* adjacentVertices_runLength, double* vertices, double* edgeLengths);
__device__ int getV0FromRunLength(int numVertices, int av, int* adjacentVertices_runLength);
__device__ double cuda_l2norm_diff(int vi, int v0, double* vertices);
__global__ void getMinEdgeLength(int numAdjacentVertices, int numVertices, int* adjacentVertices_runLength, double* vertices, double* edgeLengths, double* minEdgeLength);
__global__ void getFPrimes(int numAdjacentVertices, int numVertices, int* flat_adjacentVertices, int* adjacentVertices_runLength, double* featureVectors, double* minEdgeLength, double* vertices, double* f_primes);
__global__ void getCircleSectors(
	int numVertices, 
	int* adjacentVertices_runLength,
	int* facesOfVertices_runLength, 
	int* flat_facesOfVertices, 
	int* flat_adjacentVertices,
	int* faces, 
	double* minEdgeLength, 
	double* featureVectors, 
	double* edgeLengths,
	double* circleSectors
);
__device__ void getViAndVip1FromV0andFi(int v0, int fi, int* faces, int& vi, int& vip1);
__device__ double getEdgeLengthOfV0AndVi(int v0, int vi, int* adjacentVertices_runLength, int* flat_adjacentVertices, double* edgeLengths);

int main(){
	/*************************************************************************/
	std::cout << std::endl << "****** Initializing CUDA..." << std::endl;
	/*************************************************************************/
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query...\n");
	if(devCount <= 0)
		std::cout << "No CUDA devices found." << std::endl;
	else
		printCUDAProps(devCount);
	int blockSize;
	int numBlocks;
	/*************************************************************************/
	std::cout << "****** CUDA Initialized." << std::endl;
	/*************************************************************************/



	/*************************************************************************/
	std::cout << std::endl << "****** Loading Mesh..." << std::endl;
	/*************************************************************************/
	int numVertices;
	int numFaces;
	double* vertices;
	double* featureVectors;
	int* faces;

	loadMesh_ply("../example_meshes/h.ply", numVertices, &vertices, &featureVectors, numFaces, &faces);
	//printMesh(numVertices, vertices, featureVectors, numFaces, faces);
	/*************************************************************************/
	std::cout << "****** Finished Loading." << std::endl;
	/*************************************************************************/


	
	/*************************************************************************/
	std::cout << std::endl << "****** Begin Building Tables..." << std::endl;
	/*************************************************************************/
	std::cout << "Building table of faces by vertex, " << std::endl;
	std::cout << "and table of adjacent vertices by vertex..." << std::endl;
	std::set<int> facesOfVertices[numVertices] = {};
	std::set<int> adjacentVertices[numVertices] = {};
	
	//int numCombos = numFaces * numVertices;
	//int blockSize = 256;
	//int numBlocks = (numCombos + blockSize - 1) / blockSize;
	//buildLookupTables<<<numBlocks, blockSize>>>(numFaces, faces, facesOfVertices, adjacentVertices);

	std::cout << "Iterating over each face as f..." << std::endl;
	//TODO: Determine if this way is optimal:
	//	edges saved twice, once in each direction, but enables use of runLength array...	

	for(int f = 0; f < numFaces; f++){
		for(int i = 0; i < 3; i++){ //TODO: relies on there always being 3 vertices to a face
			int a = f*3+(i+0)%3;
			int b = f*3+(i+1)%3;
			int c = f*3+(i+2)%3;
			int v = faces[a];
			facesOfVertices[v].insert(f);
			adjacentVertices[v].insert(faces[b]);
			adjacentVertices[v].insert(faces[c]);
		}
	}
	
	/*// Print facesOfVertices
	for(int v = 0; v < numVertices; v++){
		std::cerr << "facesOfVertices[" << v << "] ";
		for(int elem : facesOfVertices[v])
			std::cerr << elem << " ";
		std::cerr << std::endl;
	}
	// Print adjacentVertices
	for(int v = 0; v < numVertices; v++){
		std::cerr << "adjacentVertices[" << v << "] ";
		for(int elem : adjacentVertices[v])
			std::cerr << elem << " ";
		std::cerr << std::endl;
	}*/
	
	// Determine runlengths of adjacentVertices and facesofVertices
	int* adjacentVertices_runLength;
	int* facesOfVertices_runLength;
	hipMallocManaged(&adjacentVertices_runLength, numVertices*sizeof(int));
	hipMallocManaged(&facesOfVertices_runLength,  numVertices*sizeof(int));
	adjacentVertices_runLength[0] = adjacentVertices[0].size();
	facesOfVertices_runLength[0]  = facesOfVertices[0].size();
	std::cout << "Iterating over each vertex as v0..." << std::endl;
	for(int v0 = 0+1; v0 < numVertices; v0++){
		adjacentVertices_runLength[v0] = adjacentVertices_runLength[v0-1] + adjacentVertices[v0].size();
		facesOfVertices_runLength[v0]  = facesOfVertices_runLength[v0-1]  + facesOfVertices[v0].size();
	}
	
	// Flatten adjacentVerticies and facesOfVertices
	int numAdjacentVertices = adjacentVertices_runLength[numVertices-1];
	int numFacesOfVertices  = facesOfVertices_runLength[numVertices-1];
	int* flat_adjacentVertices;
	int* flat_facesOfVertices;
	hipMallocManaged(&flat_adjacentVertices, numAdjacentVertices*sizeof(int));
	hipMallocManaged(&flat_facesOfVertices, numFacesOfVertices*sizeof(int));
	int r = 0;
	int s = 0;
	std::cout << "Iterating over each vertex as v0..." << std::endl;
	for(int v0 = 0; v0 < numVertices; v0++){
		for(std::set<int>::iterator vi_iter = adjacentVertices[v0].begin(); vi_iter != adjacentVertices[v0].end(); vi_iter++){
			int vi = *vi_iter;
			flat_adjacentVertices[r] = vi;
			//std::cout << "flat_adjacentVertices[" << r << "] " << flat_adjacentVertices[r] << std::endl;
			r++;
		}
		for(std::set<int>::iterator vi_iter = facesOfVertices[v0].begin(); vi_iter != facesOfVertices[v0].end(); vi_iter++){
			int vi = *vi_iter;
			flat_facesOfVertices[s] = vi;
			//std::cout << "flat_facesOfVertices[" << s << "] " << flat_facesOfVertices[s] << std::endl;
			s++;
		}
	}
	
	// Precalculate Edge Lengths
	double* edgeLengths;
	hipMallocManaged(&edgeLengths, numAdjacentVertices*sizeof(double));
	blockSize = 32;
	numBlocks = max(1, numAdjacentVertices / blockSize);
	std::cout << "getEdgeLengths<<<" << numBlocks << ", " << blockSize <<">>(" << numAdjacentVertices << ")" << std::endl;
	getEdgeLengths<<<numBlocks, blockSize>>>(numAdjacentVertices, numVertices, flat_adjacentVertices, adjacentVertices_runLength, vertices, edgeLengths);
	hipDeviceSynchronize();	//wait for GPU to finish before accessing on host
	/*************************************************************************/
	std::cout << "****** Finished Building Tables." << std::endl;
	/*************************************************************************/



	/*************************************************************************/
	std::cout << std::endl << "****** Begin Calculating..." << std::endl;
	/*************************************************************************/
	std::cout << "Calculating minimum edge length among adjacent vertices..." << std::endl;
	double* minEdgeLength;
	hipMallocManaged(&minEdgeLength, numVertices*sizeof(double));
	blockSize = 8;
	numBlocks = max(1, numVertices / blockSize);
	std::cout << "getMinEdgeLength<<<" << numBlocks << ", " << blockSize << ">>(" << numVertices << ")" << std::endl;
	getMinEdgeLength<<<numBlocks, blockSize>>>(numAdjacentVertices, numVertices, adjacentVertices_runLength, vertices, edgeLengths, minEdgeLength);
	hipDeviceSynchronize();

	std::cout << std::endl << "Calculating f', weighted mean f0 and fi by distance..." << std::endl;
	double* f_primes;
	hipMallocManaged(&f_primes, numAdjacentVertices*sizeof(double));
	blockSize = 32;
	numBlocks = max(1, numAdjacentVertices / blockSize);
	std::cout << "getFPrimes<<<" << numBlocks << ", " << blockSize << ">>(" << numAdjacentVertices << ")" << std::endl;
	getFPrimes<<<numBlocks, blockSize>>>(numAdjacentVertices, numVertices, flat_adjacentVertices, adjacentVertices_runLength, featureVectors, minEdgeLength, vertices, f_primes);
	hipDeviceSynchronize();
	
	std::cout << std::endl << "Calculating circle_sectors..." << std::endl;
	double* circleSectors;
	hipMallocManaged(&circleSectors, numVertices*sizeof(double));
	blockSize = 8;
	numBlocks = max(1, numVertices / blockSize);
	std::cout << "getCircleSectors<<<" << numBlocks << ", " << blockSize << ">>(" << numVertices << ")" << std::endl;
	getCircleSectors<<<numBlocks, blockSize>>>(
		numVertices, 
		adjacentVertices_runLength, 
		facesOfVertices_runLength, 
		flat_facesOfVertices, 
		flat_adjacentVertices, 
		faces, 
		minEdgeLength, 
		featureVectors, 
		edgeLengths, 
		circleSectors
	);
	hipDeviceSynchronize();
	/*************************************************************************/
	std::cout << "****** Finished Calculating." << std::endl;
	/*************************************************************************/
}

void printCUDAProps(int devCount){
	printf("There are %d CUDA devices.\n", devCount);

	// Iterate through devices
	for (int i = 0; i < devCount; ++i)
	{
		// Get device properties
		printf("\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		
    	printf("Name:                          %s\n",  devProp.name);
    	printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    	printf("Clock rate:                    %d\n",  devProp.clockRate);
    	printf("Total constant memory:         %u\n",  devProp.totalConstMem);
		printf("CUDA Capability Major/Minor version number:    %d.%d\n", devProp.major, devProp.minor);

	}
}

void loadMesh_ply(std::string fileName, int& numVertices, double** vertices, double** featureVectors, int& numFaces, int** faces){
	bool inHeaderSection = true;
	int faceSectionBegin;
	int vi = 0;
	int fi = 0;
	
	int v_idx = 0;
	int x_idx;
	int y_idx;
	int z_idx;

	std::ifstream infile(fileName);

	// read every line in the file
	std::string line;
	int lineNumber = 0;
	while(std::getline(infile, line)){
		// 3 sections: header, vertices, faces
		if(inHeaderSection){
			// parse for numVertices and numFaces
			if(line.substr(0, 7) == "element"){
				if(line.substr(8, 6) == "vertex"){
					std::vector<std::string> words = split<std::string>(line);
					std::istringstream convert(words[2]);
					convert >> numVertices;
				}else if(line.substr(8, 4) == "face"){
					std::vector<std::string> words = split<std::string>(line);
					std::istringstream convert(words[2]);
					convert >> numFaces;
				}
			// parse for coord indexes
			}else if(line.substr(0, 8) == "property"){
				std::vector<std::string> words = split<std::string>(line);
				if(words[2] == "x")
					x_idx = v_idx;
				else if(words[2] == "y")
					y_idx = v_idx;
				else if(words[2] == "z")
					z_idx = v_idx;
				v_idx++;
			}else if(line.substr(0, 10) == "end_header"){
				inHeaderSection = false;
				faceSectionBegin = lineNumber + 1 + numVertices;
				//(*vertices) = (double*) malloc(3 * numVertices * sizeof(double));
				//(*faces) = (int*) malloc(3 * numFaces * sizeof(int));
				hipMallocManaged(&(*vertices), 3 * numVertices * sizeof(double));
				hipMallocManaged(&(*featureVectors), numVertices * sizeof(double));
				hipMallocManaged(&(*faces), 3 * numFaces * sizeof(int));
			}
		}else if(lineNumber < faceSectionBegin){
			std::vector<double> coords = split<double>(line);
			(*vertices)[vi*3 + 0] = coords[x_idx];
			(*vertices)[vi*3 + 1] = coords[y_idx];
			(*vertices)[vi*3 + 2] = coords[z_idx];
			//TODO: Are feature vectors stored in PLY file?
			(*featureVectors)[vi] = 1;
			vi++;
		}else{
			std::vector<int> coords = split<int>(line);
			(*faces)[fi*3 + 0] = coords[1]; //coords[0] is list size
			(*faces)[fi*3 + 1] = coords[2];
			(*faces)[fi*3 + 2] = coords[3];
			fi++;
		}
		lineNumber++;
	}
}

void printMesh(
	int numVertices, 
	double* vertices, 
	double* featureVectors, 
	int numFaces, 
	int* faces
){
	for(int v = 0; v < numVertices; v++){
		std::cout << "vertices[" << v << "] = ";
		for(int i=0; i < 3; i++){
			if(i > 0)
				std::cout << ", ";
			std::cout << vertices[v*3+i];
		}
		std::cout << " featureVector = " << featureVectors[v] << std::endl;
	}
	for(int f = 0; f < numFaces; f++)
		std::cout << f << " = {" << faces[f*3+0] << ", " << faces[f*3+1] << ", " << faces[f*3+2] << "}" <<std::endl;
}


__global__
void buildLookupTables(int numFaces, int* faces, int* facesOfVertices, int* adjacentVertices){
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//int stride = blockDim.x * gridDim.x;

	//int v = index / numFaces;
	//int f = index % numFaces;
}

__global__
void getEdgeLengths(int numAdjacentVertices, int numVertices, int* flat_adjacentVertices, int* adjacentVertices_runLength, double* vertices, double* edgeLengths){
	//TODO Optimization analysis: storage vs speed
	//this:
	//	flat_adjacentVertices = 6nV
	//	adjacentVertices_runLength = 1nV
	//	index search requires averagePairCount per Vertex (6nV)
	//fully indexed:
	//	flat_adjacentVertices = 3*6nV (can be halved if redundant AVs are not stored)
	//	no runLength required
	//	no index search time
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x; //0-95
	int stride = blockDim.x * gridDim.x; //32*3 = 96

	// Use all availble threads to do all numAdjacentVertices
	for(int av = global_threadIndex; av < numAdjacentVertices; av += stride){
		int vi = flat_adjacentVertices[av];
		int v0 = getV0FromRunLength(numVertices, av, adjacentVertices_runLength);
		edgeLengths[av] = cuda_l2norm_diff(vi, v0, vertices);
		//printf("edgeLength[%d]\t(v0 %d, vi %d)\t%g\n", av, v0, vi, edgeLengths[av]);
	}
}

__device__
int getV0FromRunLength(int numVertices, int av, int* adjacentVertices_runLength){
	//TODO: measure performance	
	//this: 
	//	pros, smaller memory, 
	//	cons, need this loop to determine v0! (do intelligent search instead)
	//alternatively: save v0 as a second value per index of flat_adjacentVertices
	//	pros, v0 is always known
	//	cons flat_adjacentVertices doubles in size
	int v0;
	for(int v = 0; v < numVertices; v++){
		if(av < adjacentVertices_runLength[v]){
			//printf("[%d, %d, %d, %d]:", blockIndex, local_threadIndex, global_threadIndex, av);
			v0 = v;
			break;
		}
	}
	return v0;
}

__device__
double cuda_l2norm_diff(int vi, int v0, double* vertices){
	return sqrt((double) (vertices[(vi*3)+0] - vertices[(v0*3)+0])*(vertices[(vi*3)+0] - vertices[(v0*3)+0])
					   + (vertices[(vi*3)+1] - vertices[(v0*3)+1])*(vertices[(vi*3)+1] - vertices[(v0*3)+1])
					   + (vertices[(vi*3)+2] - vertices[(v0*3)+2])*(vertices[(vi*3)+2] - vertices[(v0*3)+2]));
}

__global__
void getMinEdgeLength(int numAdjacentVertices, int numVertices, int* adjacentVertices_runLength, double* vertices, double* edgeLengths, double* minEdgeLength){
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	// Use all availble threads to do all numVertices as v0
	for(int v0 = global_threadIndex; v0 < numVertices; v0 += stride){
		int av_begin = (v0 == 0 ? 0 : adjacentVertices_runLength[v0-1]);
		for(int av = av_begin; av < adjacentVertices_runLength[v0]; av++){
			if(minEdgeLength[v0] <= 0 || edgeLengths[av] <= minEdgeLength[v0]){
				minEdgeLength[v0] = edgeLengths[av];
			}
		}
		//printf("minEdgeLength[%d] %f\n", v0, minEdgeLength[v0]);
	}
}

__global__
void getFPrimes(int numAdjacentVertices, int numVertices, int* flat_adjacentVertices, int* adjacentVertices_runLength, double* featureVectors, double* minEdgeLength, double* vertices, double* f_primes){
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x; //0-95
	int stride = blockDim.x * gridDim.x; //32*3 = 96
	for(int av = global_threadIndex; av < numAdjacentVertices; av += stride){
		int vi = flat_adjacentVertices[av];
		int v0 = getV0FromRunLength(numVertices, av, adjacentVertices_runLength);
		f_primes[av] = featureVectors[v0] + minEdgeLength[v0] * (featureVectors[vi] - featureVectors[v0]) / cuda_l2norm_diff(vi, v0, vertices);
		//printf("f_primes[%d]\t(v0 %d, vi %d)\t%g\n", av, v0, vi, f_primes[av]);
	}
}

__global__
void getCircleSectors(
	int numVertices, 
	int* adjacentVertices_runLength,
	int* facesOfVertices_runLength, 
	int* flat_facesOfVertices, 
	int* flat_adjacentVertices,
	int* faces, 
	double* minEdgeLength, 
	double* featureVectors, 
	double* edgeLengths,
	double* circleSectors
){
	int global_threadIndex = blockIdx.x * blockDim.x + threadIdx.x; //0-95
	int stride = blockDim.x * gridDim.x; //32*3 = 96

	double accuFuncVals = 0.0;
	double accuArea = 0.0;

	// Use all availble threads to do all numVertices as v0
	for(int v0 = global_threadIndex; v0 < numVertices; v0 += stride){
		int fi_begin = (v0 == 0 ? 0 : facesOfVertices_runLength[v0-1]);
		for(int fi = fi_begin; fi < facesOfVertices_runLength[v0]; fi++){
			//currFace->getFuncVal1RingSector( this, rMinDist, currArea, currFuncVal ); //ORS.307
				//get1RingSectorConst();
				int vi, vip1;
				getViAndVip1FromV0andFi(v0, flat_facesOfVertices[fi], faces, vi, vip1);
				//printf("[%d]\t[%d]\t%d\t%d\n", v0, flat_facesOfVertices[fi], vi, vip1);

				//TODO: Ensure edges A, B, C are correct with v0, vi, vip1; also regarding funcVals later
				//ORS.456
				double lengthEdgeA = getEdgeLengthOfV0AndVi(vi, vip1, adjacentVertices_runLength, flat_adjacentVertices, edgeLengths);
				double lengthEdgeB = getEdgeLengthOfV0AndVi(v0, vip1, adjacentVertices_runLength, flat_adjacentVertices, edgeLengths);
				double lengthEdgeC = getEdgeLengthOfV0AndVi(v0, vi,   adjacentVertices_runLength, flat_adjacentVertices, edgeLengths);
				double alpha = acos( ( lengthEdgeB*lengthEdgeB + lengthEdgeC*lengthEdgeC - lengthEdgeA*lengthEdgeA ) / ( 2*lengthEdgeB*lengthEdgeC ) );

				double rNormDist = minEdgeLength[v0];
				double lenCenterToA = lengthEdgeC;
				double lenCenterToB = lengthEdgeB;
			
				//ORS.403 Area - https://en.wikipedia.org/wiki/Circular_sector#Area
				//*changed from m to r to skip "passthrough" see ORS.372
				double rSectorArea = rNormDist * rNormDist * alpha / 2.0; // As alpha is already in radiant.

				//ORS.412 Function values interpolated f'_i and f'_{i+1}
				// Compute the third angle using alpha/2.0 and 90°:
				double beta = ( M_PI - alpha ) / 2.0;
				// Law of sines
				double diameterCircum = rNormDist / sin( beta ); // Constant ratio equal longest edge

				//ORS.420 Distances for interpolation
				double mRatioCA = diameterCircum / lenCenterToA;
				double mRatioCB = diameterCircum / lenCenterToB;
				// Circle segment, center of gravity - https://de.wikipedia.org/wiki/Geometrischer_Schwerpunkt#Kreisausschnitt
				double mCenterOfGravityDist = ( 2.0 * sin( alpha ) ) / ( 3.0 * alpha );

				//ORS.357 Fetch function values
				double funcValCenter = featureVectors[v0];
				double funcValA = featureVectors[vi];
				double funcValB = featureVectors[vip1];

				//ORS.365 Interpolate
				double funcValInterpolA = funcValCenter*(1.0-mRatioCA) + funcValA*mRatioCA;
				double funcValInterpolB = funcValCenter*(1.0-mRatioCB) + funcValB*mRatioCB;

				//ORS.369 Compute average function value at the center of gravity of the circle sector
				double rSectorFuncVal = funcValCenter*( 1.0 - mCenterOfGravityDist ) +
								 ( funcValInterpolA + funcValInterpolB ) * mCenterOfGravityDist / 2.0;

			double currFuncVal = rSectorFuncVal;
			double currArea = rSectorArea;
			
			//ORS.309
			accuFuncVals += currFuncVal * currArea;
			accuArea += currArea;
		}

		circleSectors[v0] = accuFuncVals / accuArea;
		printf("circleSectors[%d] %f\n", v0, circleSectors[v0]);
	}
}

__device__
void getViAndVip1FromV0andFi(int v0, int fi, int* faces, int& vi, int& vip1){
	//printf("faces[%d*3+{0,1,2}] {%d, %d, %d}\n", fi, faces[(fi*3)+0], faces[(fi*3)+1], faces[(fi*3)+2]);
	bool isViAssigned = false;
	for(int i = 0; i < 3; i++){ // for each vertex in this face (a, b, c)
		int v = faces[fi*3+i];
		if(v != v0){ // exclude v0
			if(isViAssigned){
				vip1 = v; // assign the other corner to vip1
			}else{
				vi = v; // assign the first corner to vi
				isViAssigned = true;
			}
		}
	}
}


__device__
double getEdgeLengthOfV0AndVi(int v0, int vi, int* adjacentVertices_runLength, int* flat_adjacentVertices, double* edgeLengths){
	//TODO: Error handling?
	int av_begin = (v0 == 0 ? 0 : adjacentVertices_runLength[v0-1]);
	double edgeLength;
	for(int av = av_begin; av < adjacentVertices_runLength[v0]; av++){
		if(flat_adjacentVertices[av] == vi){
			edgeLength = edgeLengths[av];
			break;
		}
	}
	return edgeLength;
}


